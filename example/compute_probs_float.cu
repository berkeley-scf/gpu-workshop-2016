
#include <hip/hip_runtime.h>
extern "C"
__global__ void compute_probs(float* alphas, float* rands, float* probs, int n, int K, int M) {
    // assign overall id/index of the thread = id of row
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if(i < n) {
      float maxval;    
      int m, k;
      int maxind;
      float M_d = (float) M; 
      float* w = new float[K];

      for(k = 0; k < K; ++k){   // initialize probs (though already done on CPU)
         probs[i*K + k] = 0.0;
      }
      for(m = 0; m < M; ++m){   // loop over Monte Carlo iterations
        for(k = 0; k < K; ++k){  // generate W ~ N(alpha, 1)
          w[k] = alphas[i*K + k] + rands[m*K + k];
        }

        // determine which category has max W
        maxind = K-1;
        maxval = w[K-1];
        for(k = 0; k < (K-1); ++k){
          if(w[k] > maxval){
            maxind = k;
            maxval = w[k];
          } 
        }
        probs[i*K + maxind] += 1.0;
      }
      // compute final proportions
      for(k = 0; k < K; ++k) {
        probs[i*K + k] /= M_d;
      }
      free(w);
    }

}
