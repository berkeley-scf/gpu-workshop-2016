
#include <hip/hip_runtime.h>
extern "C"
__global__ void compute_probs(double* alphas, double* rands, double* probs, int n, int K, int M) {
    // assign overall id/index of the thread = id of row
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if(i < n) {
      double maxval;    
      int m, k;
      int maxind;
      double M_d = (double) M; 
      double* w = new double[K];

      for(k = 0; k < K; ++k){  // initialize probs (though already done on CPU)
         probs[k*n + i] = 0.0;
      }

      // core computations
      for(m = 0; m < M; ++m){    // loop over Monte Carlo iterations
        for(k = 0; k < K; ++k){  // generate W ~ N(alpha, 1)
          // with +i we now have unit strides in inner loop
          w[k] = alphas[k*n + i] + rands[k*M + m];
        }

        // determine which category has max W
        maxind = K-1;
        maxval = w[K-1];
        for(k = 0; k < (K-1); ++k){
          if(w[k] > maxval){
            maxind = k;
            maxval = w[k];
          } 
        }
        probs[maxind*n + i] += 1.0;
      }

      // compute final proportions
      for(k = 0; k < K; ++k) {
        // unit strides
        probs[k*n + i] /= M_d;
      }
      free(w);
    }

}
