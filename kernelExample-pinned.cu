#include <stdlib.h>
#include <sys/time.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define SQRT_TWO_PI 2.506628274631000
#define BLOCK_D1 1024
#define BLOCK_D2 1
#define BLOCK_D3 1

// Note: Needs compute capability >= 2.0 for calculation with doubles, so compile with:
// nvcc kernelExample-pinned.cu -arch=compute_20 -code=sm_20,compute_20 -o kernelExample-pinned
// -use_fast_math

// CUDA kernel:
__global__ void calc_loglik(double* vals, int n, double mu, double sigma) {
   // note that this assumes no third dimension to the grid
    // id of the block
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    // size of each block (within grid of blocks)
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    // id of thread in a given block
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    // assign overall id/index of the thread
    int idx = myblock * blocksize + subthread;

        if(idx < n) {
            double std = (vals[idx] - mu)/sigma;
            double e = exp( - 0.5 * std * std);
            vals[idx] = e / ( sigma * SQRT_TWO_PI);
        }
}

int calc_loglik_cpu(double* vals, int n, double mu, double sigma) {
  double std, e;
  for(int idx = 0; idx < n; idx++) {
    std = (vals[idx] - mu)/sigma;
    e = exp( - 0.5 * std * std);
    vals[idx] = e / ( sigma * SQRT_TWO_PI);
  }
  return 0;
}


/* --------------------------- host code ------------------------------*/
void fill( double *p, int n ) {
  int i;
  srand48(0);
  for( i = 0; i < n; i++ )
    p[i] = 2*drand48()-1;
}

double read_timer() {
  struct timeval end;
  gettimeofday( &end, NULL );
  return end.tv_sec+1.e-6*end.tv_usec;
}

int main (int argc, char *argv[]) {
  double* cpu_vals;
  double* gpu_vals;
  int n;
  hipError_t cudaStat;
 
  printf("====================================================\n");
  for( n = 32768; n <= 134217728; n*=8 ) {
    // allocated pinned and mapped memory on CPU
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc((void**)&cpu_vals, n*sizeof(double), hipHostMallocMapped);

    // map the CPU storage to the GPU to the CPU storage
    cudaStat = hipHostGetDevicePointer(&gpu_vals, cpu_vals, 0);
    if(cudaStat != hipSuccess) {
      printf ("device memory mapping failed");
      return EXIT_FAILURE;
    }

    const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
    
    int tmp = ceil(pow(n/BLOCK_D1, 0.5));
    printf("Grid dimension is %i x %i\n", tmp, tmp);
    dim3 gridSize(tmp, tmp, 1);

    int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*tmp*tmp;
    if (nthreads < n){
        printf("\n============ NOT ENOUGH THREADS TO COVER n=%d ===============\n\n",n);
    } else {
        printf("Launching %d threads (n=%d)\n", nthreads, n);
    }

    double mu = 0.0;
    double sigma = 1.0;

    // simulate 'data'
    fill(cpu_vals, n);
    printf("Input values: %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    hipDeviceSynchronize();
    double tInit = read_timer();

    // do the calculation
    calc_loglik<<<gridSize, blockSize>>>(gpu_vals, n, mu, sigma);
    
    hipDeviceSynchronize();
    double tCalc = read_timer();

    printf("Output values: %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    // do calculation on CPU for comparison (unfair as this will only use one core)
    fill(cpu_vals, n);
    double tInit2 = read_timer();
    calc_loglik_cpu(cpu_vals, n, mu, sigma);
    double tCalcCPU = read_timer();

    printf("Output values (CPU): %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    printf("Timing results for n = %d\n", n);
    printf("Calculation time (GPU): %f\n", tCalc - tInit);
    printf("Calculation time (CPU): %f\n", tCalcCPU - tInit2);

    printf("Freeing memory...\n");
    printf("====================================================\n");
    hipHostFree(cpu_vals);

  }
  printf("\n\nFinished.\n\n");
  return 0;
}

